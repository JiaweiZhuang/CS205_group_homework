#include "hip/hip_runtime.h"
# include "../shared/timing.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>

# include <hip/hip_runtime.h>
# include "hipblas.h"

int main() {
    // set up problem size
    int N = pow(2, 10);
    int size = N * N;

    // for CUBLAS
    double alpha=1.0, beta=0.0;
    hipblasHandle_t handle = 0;

    // malloc host memory and initialize data
    double* h_A = (double*)malloc(sizeof(double) * size);
    double* h_B = (double*)malloc(sizeof(double) * size);
    double* h_C = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; i++) {
        h_A[i] = 1.0;
        h_B[i] = 1.0;
    }

    // start timing
    double iStart, iElaps;
    iStart = seconds();

    // Create the cuBLAS handle
    hipblasCreate(&handle);

    // malloc device global memory 
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

	// Transfer inputs to the device
	hipblasSetMatrix(N, N, sizeof(double), h_A, N, d_A, N);
	hipblasSetMatrix(N, N, sizeof(double), h_B, N, d_B, N);

    // Execute the matrix-matrix multiplication
    // C = alpha*A*B + beta*C
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, 
                &alpha, d_A, N, d_B, N, 
                &beta,  d_C, N); 

	// Retrieve the output matrix from the device	
	hipblasGetMatrix(N, N, sizeof(double), d_C, N, h_C, N);

    // end of timing
    iElaps = seconds() - iStart;

    // print the results
    printf("First element =  %f \n", h_C[0]);
    printf("Time elapse = %f ms \n", iElaps*1000.0);

    // clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}
