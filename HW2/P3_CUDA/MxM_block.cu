#include "hip/hip_runtime.h"
# include "../shared/timing.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# define BSIZE 16

// device code for block MxM
// Omit the d_ prefix because all variables are on device
__global__ void MxM_block(double* A, double* B, double* C, const int N) {

    // store sub-matrix in the shared memory
    // They should be in the L1 cache of each streaming processor
    __shared__ double A_tile[BSIZE][BSIZE];
    __shared__ double B_tile[BSIZE][BSIZE];
    __shared__ double C_tile[BSIZE][BSIZE];
   
    // each thread computes one element of the block sub-matrix
    // initialize the element to 0
    C_tile[threadIdx.y][threadIdx.x] = 0.0;

    // pseudo code:
    // C[i1:i2][j1:j2] += A[i1:i2][k1:k2] * B[k1:k2][j1:j2]
    // For each block, i,j is known:
    // (i ranges from i1 to i2, j ranges from j1 to j2)
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // Only k is changing, so we loop over k:
    // each iteration (k1,k2) -> (k1+BSIZE,k2+BSIZE)
    for (int tileIdx = 0; tileIdx < N/BSIZE; tileIdx++){

        // Load the matrices from global memory to shared memory
        // each thread loads one element
		// note that we use 1D index for global matrix but 2D index for a tile
        int k_A = tileIdx*BSIZE+threadIdx.x; // k is the row index for A 
        int k_B = tileIdx*BSIZE+threadIdx.y; // k is the column index for B
        A_tile[threadIdx.y][threadIdx.x] = A[i*N + k_A];
        B_tile[threadIdx.y][threadIdx.x] = B[k_B*N + j];

        // Synchronize to make sure the entire tile is loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < BSIZE; k++) {
            C_tile[threadIdx.y][threadIdx.x] += 
            A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

	// write back to global memory
    // each thread writes one element
    C[i*N + j] = C_tile[threadIdx.y][threadIdx.x];

    // Final note:
    // It seems that using C_temp (a scalar)
    // instead of C_tile[threadIdx.y, threadIdx.x]
    // is more memory-efficient.
    // because C_temp can be stored in the register of each thread.
    // So we can save more shared memory for A_tile and B_tile
    // But to follow the homework instruction we still allocate
    // shared memory for C_tile 

}


// host code is almost the same as the naive one
int main() {
    // set up problem size
    int N = pow(2, 12);
    int size = N * N;

    // malloc host memory and initialize data
    double* h_A = (double*)malloc(sizeof(double) * size);
    double* h_B = (double*)malloc(sizeof(double) * size);
    double* h_C = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; i++) {
        h_A[i] = 1.0;
        h_B[i] = 1.0;
    }

	// malloc device global memory and transfer data from host to device
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    hipMemcpy(d_A, h_A, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(double), hipMemcpyHostToDevice);

	// Invoke kernel at host side
	// One thread-block calculates on block of matrix C
    // Inside the block, one thread handles one element
    dim3 dimBlock(BSIZE, BSIZE);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

    // warm-up. The first call is significantly slower than the following calls.
    MxM_block<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
	// start timing and execute the kernel function
    double iStart, iElaps;
    iStart = seconds();
    MxM_block<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    iElaps = seconds() - iStart;

    // copy kernel result back to host side
    hipMemcpy(h_C, d_C, size*sizeof(double), hipMemcpyDeviceToHost);

	// print the results
    printf("First element =  %f \n", h_C[0]);
    printf("Time elapse = %f ms \n", iElaps*1000.0);
	
    // clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return(0);
   
}
