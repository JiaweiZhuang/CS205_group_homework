#include "hip/hip_runtime.h"
# include "../shared/timing.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>

// device code for naive MxM
// Omit the d_ prefix because all variables are on device
// We don't make use of shared memory (for each block) in this naive version
__global__ void MxM_naive(double* A, double* B, double* C, const int N) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;    // Row i of matrix C
    int j = blockIdx.x * blockDim.x + threadIdx.x;    // Column j of matrix C

    double C_temp = 0;
    for (int k=0; k<N; k++) {
        // use 1D indexing
        C_temp += A[i*N + k] * B[k*N + j];
    }

	// write back to global memory
	// no synchronization needed here because one thread handles one element
    C[i*N + j] = C_temp;

}


int main() {
    // set up problem size
    int N = pow(2, 12);
    int size = N * N;

    // malloc host memory and initialize data
    double* h_A = (double*)malloc(sizeof(double) * size);
    double* h_B = (double*)malloc(sizeof(double) * size);
    double* h_C = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; i++) {
        h_A[i] = 1.0;
        h_B[i] = 1.0;
    }

	// malloc device global memory and transfer data from host to device
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    hipMemcpy(d_A, h_A, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(double), hipMemcpyHostToDevice);

	// Invoke kernel at host side
	// One thread calculates one element of matrix C.
	// There is limit (512 or 1024) on the number of threads in a block,
 	// so we use many blocks to generate N*N threads in total.
	// The 'block' here has nothing to do with the block matrix multiplication,
	// because block ids are immediately converted back to matrix indices.
    dim3 dimBlock(16, 16);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

    // warm-up. The first call is significantly slower than the following calls.
    MxM_naive<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
	// start timing and execute the kernel function
    double iStart, iElaps;
    iStart = seconds();
    MxM_naive<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    iElaps = seconds() - iStart;

    // copy kernel result back to host side
    hipMemcpy(h_C, d_C, size*sizeof(double), hipMemcpyDeviceToHost);

	// print the results
    printf("First element =  %f \n", h_A[0]);
    printf("First element =  %f \n", h_C[0]);
    printf("Time elapse = %f ms \n", iElaps*1000.0);
	
    // clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return(0);
   
}
