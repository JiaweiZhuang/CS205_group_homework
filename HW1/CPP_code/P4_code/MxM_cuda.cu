
#include <hip/hip_runtime.h>
# include <stdlib.h>
# include <stdio.h>
# include <time.h>
# include <math.h>
# define BSIZE 32

// Forward declaration of the device multiplication function
__global__ void blockMxM(double*, double*, double*, int);
// Host multiplication function
double* cuda_MxM(double* A, double* B, int N) {
    double* AA;
    double* BB;
    double* CC;
    int size = N * N;

    // Put into device
    hipMalloc((void **)&AA, size * sizeof(double));
    hipMemcpy(AA, A, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void **)&BB, size * sizeof(double));
    hipMemcpy(BB, B, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void **)&CC, size * sizeof(double));

    // Each device is responsible for one block
    dim3 dimBlock(BSIZE, BSIZE);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);

    blockMxM<<<dimGrid, dimBlock>>>(AA, BB, CC, N);

    // Read result from the device
    double* C = (double*)malloc(size * sizeof(double));
    
    hipMemcpy(C, CC, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(AA);
    hipFree(BB);
    // cudaFree(CC);
    return C;
}

// Device code for mutiplication on each block
__global__ void blockMxM(double* A, double* B, double* C, int N) {

    int stripeA = (blockIdx.y * BSIZE) * N;
    int stripeB = blockIdx.x * BSIZE;
    int blockxy = threadIdx.y * N + threadIdx.x;
    // printf("Hello from block %d, %d, thread %d, %d, sA: %d, sB: %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, stripeA, stripeB);
    
    double Csub = 0;
    
    for (int i = 0; i < (N-1) / BSIZE + 1; i++) {
        // Shared memory 
        
        __shared__ double AA[BSIZE][BSIZE];
        __shared__ double BB[BSIZE][BSIZE];

        int j = stripeA + i * BSIZE + blockxy;
        int k = stripeB + i * (BSIZE * N) + blockxy;

        AA[threadIdx.y][threadIdx.x] = A[j];
        BB[threadIdx.y][threadIdx.x] = B[k];
        __syncthreads();
        

        for (int ii = 0; ii < BSIZE; ii++) {
            Csub += AA[threadIdx.y][ii] * BB[ii][threadIdx.x];
        }
        __syncthreads();
    }
    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int l = stripeA + stripeB + blockxy;
    C[l] = Csub;
}

int main() {
    int N = pow(2, 10);
    int size = N * N;
    double* A = (double*)malloc(sizeof(double) * size);
    double* B = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; i++) {
        A[i] = 1.0;
        B[i] = 1.0;
    }
    clock_t start = clock();
    double* C = cuda_MxM(A, B, N);
    clock_t end = clock();
    int msec = (end - start) * 1000 / CLOCKS_PER_SEC;
    printf("%d milliseconds \n", msec);
    // for (int i = 0; i < size; i++) {
    //     printf("%lf\n", C[i]);
    // } 
}